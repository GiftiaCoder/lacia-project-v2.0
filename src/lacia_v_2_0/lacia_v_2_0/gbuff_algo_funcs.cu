#include "hip/hip_runtime.h"

#include "gbuff.h"
#include "gbuff_defs.cuh"

namespace lacia {
	
#define PRELU_A (0.01)
#define ELU_A (1.0)

#define SELU_A (1.0)
#define SELU_B (1.0)

#define DEF_ACTIVATE_FUNC(_name_) \
	__global__ void KERNEL_FUNC_NAME(_name_)(real value[], count calscale) { \
		CREATE_TID(calscale) \
			value[tid] = KERNEL_FUNC_NAME(_name_)(value[tid]); \
		DESTORY_TID() \
	} \
	void gbuff::_name_() { \
		CALL_KERNEL_FUNC(_name_, gpubuf(), size()); \
	}
	/****************************
	 * begin activate functions *
	 ****************************/
	__device__ real KERNEL_FUNC_NAME(tanh)(real val) {
		return tanh((double) val);
	}
	DEF_ACTIVATE_FUNC(tanh)

	__device__ real KERNEL_FUNC_NAME(tanhex)(real val) {
		return val * 0.001 + tanh((double)val);
	}
	DEF_ACTIVATE_FUNC(tanhex)

	__device__ real KERNEL_FUNC_NAME(cut)(real val) {
		return val < -1.0 ? -1.0 : (val > 1.0 ? 1.0 : val);
	}
	DEF_ACTIVATE_FUNC(cut)

	__device__ real KERNEL_FUNC_NAME(relu)(real val) {
		return val >= 0.0 ? val : 0.0;
	}
	DEF_ACTIVATE_FUNC(relu)
	
	__device__ real KERNEL_FUNC_NAME(prelu)(real val) {
		return val >= 0.0 ? val : (PRELU_A * val);
	}
	DEF_ACTIVATE_FUNC(prelu)

	__device__ real KERNEL_FUNC_NAME(elu)(real val) {
		return val >= 0.0 ? val : (ELU_A * (exp(val) - 1.0));
	}
	DEF_ACTIVATE_FUNC(elu)

	__device__ real KERNEL_FUNC_NAME(selu)(real val) {
		return SELU_B * (val >= 0.0 ? val : (SELU_A * (exp(val) - 1)));
	}
	DEF_ACTIVATE_FUNC(selu)

}
