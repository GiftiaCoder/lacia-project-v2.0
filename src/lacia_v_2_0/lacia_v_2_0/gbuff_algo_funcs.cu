#include "hip/hip_runtime.h"

#include "gbuff.h"

#include <hip/hip_runtime.h>

namespace lacia {
	
	__global__ void cualgo_tanh(real o[], real b[], count n) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < n) {
			o[tid] = tanh(b[tid]);
			tid += tnm;
		}
	}
	void gbuff::tanh() {
		cualgo_tanh << <1024, 128 >> > (gpubuf(), gpubuf(), size());
	}

	__global__ void cualgo_tanhex(real o[], real b[], count n) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < n) {
			o[tid] = 1.7159 * tanh(b[tid] * 0.6666);
			tid += tnm;
		}
	}
	void gbuff::tanhex() {
		cualgo_tanhex << <1024, 128 >> > (gpubuf(), gpubuf(), size());
	}

}
