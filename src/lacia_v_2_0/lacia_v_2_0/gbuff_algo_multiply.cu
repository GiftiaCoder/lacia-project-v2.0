#include "hip/hip_runtime.h"

#include "gbuff.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <hip/device_functions.h>

namespace lacia {

	__global__ void cualgo_dot_num(real o[], real v, int len) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < len) {
			o[tid] *= v;
			tid += tnm;
		}
	}
	void gbuff::dot(real v) {
		cualgo_dot_num << <1024, 128 >> > (gpubuf(), v, size());
	}

	__global__ void cualgo_dot_vec(real o[], real a[], int len) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < len) {
			o[tid] *= a[tid];
			tid += tnm;
		}
	}
	void gbuff::dot(gbuff &b) {
		if (size() != b.size()) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_dot_vec << <1024, 128 >> >(gpubuf(), b.gpubuf(), size());
	}

	__global__ void cualgo_dot_num(real o[], real a[], real v, int len) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < len) {
			o[tid] = a[tid] * v;
			tid += tnm;
		}
	}
	void gbuff::dot(gbuff &b, real v) {
		cualgo_dot_num<<<1024, 128>>>(gpubuf(), b.gpubuf(), v, size());
	}

	__global__ void cualgo_dot_vec(real o[], real a[], real b[], int len) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < len) {
			o[tid] = a[tid] * b[tid];
			tid += tnm;
		}
	}
	void gbuff::dot(gbuff &b1, gbuff &b2) {
		if (size() != b1.size() || size() != b2.size()) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_dot_vec<<<1024, 128>>>(gpubuf(), b1.gpubuf(), b2.gpubuf(), size());
	}

	__global__ void cualgo_cross(real o[], real a[], count group_size, real b[], count calculate_scale) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < calculate_scale) {
			o[tid] = a[tid % group_size] * b[tid / group_size];
			tid += tnm;
		}
	}
	void gbuff::cross(gbuff &b1, gbuff &b2) {
		if (b1.size() * b2.size() != size()) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_cross<<<1024, 128>>>(gpubuf(), b1.gpubuf(), b1.size(), b2.gpubuf(), b1.size() * b2.size());
	}

	__global__ void cualgo_dot_multiple(real o[], count calculate_scale, real a[], count group_size) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < calculate_scale) {
			o[tid] *= a[tid % group_size];
			tid += tnm;
		}
	}
	void gbuff::dot_multiple(gbuff &b) {
		if (size() % b.size() != 0) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_dot_multiple<<<1024, 128>>>(gpubuf(), size(), b.gpubuf(), b.size());
	}

	__global__ void cualgo_dot_multiple_t(real o[], count calculate_scale, real a[], count group_num) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < calculate_scale) {
			o[tid] *= a[tid / group_num];
			tid += tnm;
		}
	}
	void gbuff::dot_multiple_t(gbuff &b) {
		if (size() % b.size() != 0) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_dot_multiple_t<<<1024, 128>>>(gpubuf(), size(), b.gpubuf(), size() / b.size());
	}

	__global__ void cualgo_dot_multiple(real o[], real a[], count calculate_scale, real b[], count group_size) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < calculate_scale) {
			o[tid] = a[tid] * b[tid % group_size];
			tid += tnm;
		}
	}
	void gbuff::dot_multiple(gbuff &b1, gbuff &b2) {
		if (size() % b2.size() != 0 || size() != b1.size()) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_dot_multiple<<<1024, 128>>>(gpubuf(), b1.gpubuf(), b1.size(), b2.gpubuf(), b2.size());
	}

	__global__ void cualgo_dot_multiple_t(real o[], real a[], count calculate_scale, real b[], count group_num) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < calculate_scale) {
			o[tid] = a[tid] * b[tid / group_num];
			tid += tnm;
		}
	}
	void gbuff::dot_multiple_t(gbuff &b1, gbuff &b2) {
		if (size() % b2.size() != 0 || size() != b1.size()) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_dot_multiple_t<<<1024, 128>>>(gpubuf(), b1.gpubuf(), b1.size(), b2.gpubuf(), b1.size() / b2.size());
	}

}
