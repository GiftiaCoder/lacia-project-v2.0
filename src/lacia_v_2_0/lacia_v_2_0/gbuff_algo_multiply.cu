#include "hip/hip_runtime.h"

#include "gbuff.h"
#include "gbuff_defs.cuh"

namespace lacia {

	__global__ void KERNEL_FUNC_NAME(cross)(real out[], real in1[], count group_size, real in2[], count calscale) {
		CREATE_TID(calscale)
			out[tid] = in1[tid % group_size] * in2[tid / group_size];
		DESTORY_TID()
	}
	void gbuff::cross(gbuff &in1, gbuff &in2) {
		count calscale = in1.size() * in2.size();
		CHECK_ASSERT(calscale == size(), INVALID_PARAMETER_FAIL);
		CALL_KERNEL_FUNC(cross, gpubuf(), in1.gpubuf(), in1.size(), in2.gpubuf(), calscale);
	}

	__global__ void KERNEL_FUNC_NAME(grouped_multiple)(real out[], count calscale, real in[], count group_size) {
		CREATE_TID(calscale)
			out[tid] *= in[tid % group_size];
		DESTORY_TID()
	}
	void gbuff::grouped_multiple(gbuff &in) {
		CHECK_ASSERT(size() % in.size() == 0, INVALID_PARAMETER_FAIL);
		CALL_KERNEL_FUNC(grouped_multiple, gpubuf(), size(), in.gpubuf(), in.size());
	}
	__global__ void KERNEL_FUNC_NAME(grouped_multiple)(real out[], real in1[], count calscale, real in2[], count group_size) {
		CREATE_TID(calscale)
			out[tid] = in1[tid] * in2[tid % group_size];
		DESTORY_TID()
	}
	void gbuff::grouped_multiple(gbuff &in1, gbuff &in2) {
		CHECK_ASSERT((size() == in1.size() && (size() % in2.size() == 0)), INVALID_PARAMETER_FAIL);
		CALL_KERNEL_FUNC(grouped_multiple, gpubuf(), in1.gpubuf(), in1.size(), in2.gpubuf(), in2.size());
	}

	__global__ void KERNEL_FUNC_NAME(grouped_multiple_t)(real out[], count calscale, real in[], count group_num) {
		CREATE_TID(calscale)
			out[tid] *= in[tid / group_num];
		DESTORY_TID()
	}
	void gbuff::grouped_multiple_t(gbuff &in) {
		CHECK_ASSERT(size() % in.size() == 0, INVALID_PARAMETER_FAIL);
		CALL_KERNEL_FUNC(grouped_multiple_t, gpubuf(), size(), in.gpubuf(), size() / in.size());
	}
	__global__ void KERNEL_FUNC_NAME(grouped_multiple_t)(real out[], real in1[], count calscale, real in2[], count group_num) {
		CREATE_TID(calscale)
			out[tid] = in1[tid] * in2[tid / group_num];
		DESTORY_TID()
	}
	void gbuff::grouped_multiple_t(gbuff &in1, gbuff &in2) {
		CHECK_ASSERT((size() == in1.size()) && (size() % in2.size() == 0), INVALID_PARAMETER_FAIL);
		CALL_KERNEL_FUNC(grouped_multiple_t, gpubuf(), in1.gpubuf(), in1.size(), in2.gpubuf(), in1.size() / in2.size());
	}

}
