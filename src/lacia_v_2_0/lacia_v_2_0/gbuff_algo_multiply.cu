#include "hip/hip_runtime.h"

#include "gbuff.h"

#include <hip/hip_runtime.h>

namespace lacia {

	__global__ void cualgo_dot_num(real o[], real v, int len) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < len) {
			o[tid] *= v;
			tid += tnm;
		}
	}
	void gbuff::dot(real v) {
		cualgo_dot_num << <1024, 128 >> > (gpubuf(), v, size());
	}

	__global__ void cualgo_dot_vec(real o[], real a[], int len) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < len) {
			o[tid] *= a[tid];
			tid += tnm;
		}
	}
	void gbuff::dot(gbuff &b) {
		if (size() != b.size()) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_dot_vec << <1024, 128 >> >(gpubuf(), b.gpubuf(), size());
	}

	__global__ void cualgo_dot_num(real o[], real a[], real v, int len) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < len) {
			o[tid] = a[tid] * v;
			tid += tnm;
		}
	}
	void gbuff::dot(gbuff &b, real v) {
		cualgo_dot_num<<<1024, 128>>>(gpubuf(), b.gpubuf(), v, size());
	}

	__global__ void cualgo_dot_vec(real o[], real a[], real b[], int len) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < len) {
			o[tid] = a[tid] * b[tid];
			tid += tnm;
		}
	}
	void gbuff::dot(gbuff &b1, gbuff &b2) {
		if (size() != b1.size() || size() != b2.size()) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_dot_vec<<<1024, 128>>>(gpubuf(), b1.gpubuf(), b2.gpubuf(), size());
	}

	__global__ void cualgo_cross(real o[], real a[], count an, real b[], count calculate_scale) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < calculate_scale) {
			o[tid] = a[tid % an] * b[tid / an];
			tid += tnm;
		}
	}
	void gbuff::cross(gbuff &b1, gbuff &b2) {
		if (b1.size() * b2.size() != size()) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_cross<<<1024, 128>>>(gpubuf(), b1.gpubuf(), b1.size(), b2.gpubuf(), b1.size() * b2.size());
	}

	__global__ void cualgo_dot_multiple(real o[], count on, real a[], count an) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < on) {
			o[tid] *= a[tid % an];
			tid += tnm;
		}
	}
	void gbuff::dot_multiple(gbuff &b) {
		if (size() % b.size() != 0) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_dot_multiple<<<1024, 128>>>(gpubuf(), size(), b.gpubuf(), b.size());
	}

	__global__ void cualgo_dot_multiple_t(real o[], count on, real a[], count an) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < on) {
			o[tid] *= a[tid / an];
			tid += tnm;
		}
	}
	void gbuff::dot_multiple_t(gbuff &b) {
		if (size() % b.size() != 0) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_dot_multiple_t<<<1024, 128>>>(gpubuf(), size(), b.gpubuf(), b.size());
	}

	__global__ void cualgo_dot_multiple(real o[], real a[], count an, real b[], count bn) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < an) {
			o[tid] = a[tid] * b[tid % bn];
			tid += tnm;
		}
	}
	void gbuff::dot_multiple(gbuff &b1, gbuff &b2) {
		if (size() % b2.size() != 0 || size() != b1.size()) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_dot_multiple<<<1024, 128>>>(gpubuf(), b1.gpubuf(), b1.size(), b2.gpubuf(), b2.size());
	}

	__global__ void cualgo_dot_multiple_t(real o[], real a[], count an, real b[], count bn) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < an) {
			o[tid] = a[tid] * b[tid / bn];
			tid += tnm;
		}
	}
	void gbuff::dot_multiple_t(gbuff &b1, gbuff &b2) {
		if (size() % b2.size() != 0 || size() != b1.size()) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_dot_multiple_t<<<1024, 128>>>(gpubuf(), b1.gpubuf(), b1.size(), b2.gpubuf(), b2.size());
	}

}
