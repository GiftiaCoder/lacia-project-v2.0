#include "hip/hip_runtime.h"

#include "gbuff.h"

#include <hip/hip_runtime.h>

namespace lacia {

	__global__ void cualgo_set_value(real o[], count n, real v) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < n) {
			o[tid] = v;
			tid += tnm;
		}
	}
	void gbuff::zero() {
		set((real) 0.0);
	}
	void gbuff::one() {
		set((real) 1.0);
	}
	void gbuff::set(real v) {
		cualgo_set_value<<<1024, 128>>>(gpubuf(), size(), v);
	}

	__device__ real cualgo_get_rand(count seed, real min, real max) {
		count t = ((((((seed * 334379) >> 3) * 334363) >> 3) * 334349) >> 3) * 334333;
		real v = (real)t / (real)(count)0xFFFFFFFF;
		return v * (max - min) + min;

	} 
	__global__ void cualgo_set_rand(real o[], count n, real min, real max) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < n) {

			union {
				struct {
					count a;
					count b;
				};
				real *p;
			} u;
			u.p = o + tid;

			o[tid] = cualgo_get_rand(u.a ^ u.b, min, max);
			tid += tnm;
		}
	}
	void gbuff::rand(real min, real max) {
		cualgo_set_rand << <1024, 128 >> >(gpubuf(), size(), min, max);
	}

}
