#include "hip/hip_runtime.h"

#include "gbuff.h"

#include <hip/hip_runtime.h>

namespace lacia {

	__global__ void cualgo_set_value(real o[], count n, real v) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < n) {
			o[tid] = v;
			tid += tnm;
		}
	}
	void gbuff::zero() {
		set((real) 0.0);
	}
	void gbuff::one() {
		set((real) 1.0);
	}
	void gbuff::set(real v) {
		cualgo_set_value<<<1024, 128>>>(gpubuf(), size(), v);
	}

	__device__ real cualgo_get_rand(real o[], count i, real min, real max) {
		union
		{
			struct {
				count a, b;
			};
			real *p;
		} u;
		u.p = o + i;
		real v = (((((((u.a ^ u.b) * 334379) >> 2) * 334363) >> 2) * 334349) >> 2) * 334333 / (real)(count)0xFFFFFFFF;
		return v * (max - min) + min;

	} 
	__global__ void cualgo_set_rand(real o[], count n, real min, real max) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < n) {
			o[tid] = cualgo_get_rand(o, tid, min, max);
			tid += tnm;
		}
	}
	void gbuff::rand(real min, real max) {
		cualgo_set_rand << <1024, 128 >> >(gpubuf(), size(), min, max);
	}

}
