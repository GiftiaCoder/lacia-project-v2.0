#include "hip/hip_runtime.h"

#include "gbuff.h"
#include "gbuff_defs.cuh"

namespace lacia {

	__global__ void KERNEL_FUNC_NAME(set_value)(real out[], count calscale, real value) {
		CREATE_TID(calscale)
			out[tid] = value;
		DESTORY_TID()
	}
	void gbuff::zero() {
		set((real) 0.0);
	}
	void gbuff::one() {
		set((real) 1.0);
	}
	void gbuff::set(real value) {
		CALL_KERNEL_FUNC(set_value, gpubuf(), size(), value);
	}

	__device__ real KERNEL_FUNC_NAME(rand)(count seed, real min, real max) {
		count t = ((((((seed * 334379) >> 3) * 334363) >> 3) * 334349) >> 3) * 334333;
		real v = (real)t / (real)(count)0xFFFFFFFF;
		return v * (max - min) + min;

	} 
	__global__ void KERNEL_FUNC_NAME(rand)(real out[], count calscale, real min, real max, seed_t seed) {
		CREATE_TID(calscale)
			union {
				struct {
					count a;
					count b;
				};
				seed_t p;
			} u = { 0 };
			u.p = tid * sizeof(seed_t) + seed;
			out[tid] = KERNEL_FUNC_NAME(rand)(u.a ^ u.b, min, max);
		DESTORY_TID()
	}
	void gbuff::rand(real min, real max, seed_t seed) {
		if (! seed) {
			seed = (seed_t)gpubuf();
		}
		CALL_KERNEL_FUNC(rand, gpubuf(), size(), min, max, seed);
	}

}
