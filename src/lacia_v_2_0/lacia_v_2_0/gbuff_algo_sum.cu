#include "hip/hip_runtime.h"

#include "gbuff.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <hip/device_functions.h>

namespace lacia {
	
	__global__ void cualgo_cpy_sum(real o[], real b[], count group_num, count group_size) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < group_num) {
			o[tid] = b[tid * group_size];
			tid += tnm;
		}
	}

	/*
	 * if group size is even, sum_offset = sum_size
	 * else, sum_offset = sum_size + 1
	*/
	__global__ void cualgo_naive_sum(real b[], count sum_size, count sum_offset, count group_size, count calculate_scale) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < calculate_scale) {
			count idx = ((tid / sum_size) * group_size) + (tid % sum_size);
			b[idx] += b[idx + sum_offset];
			tid += tnm;
		}
	}
	static void naive_sum(real o[], real b[], count group_size, count group_num) {
		count sum_size = group_size;
		while (true) {
			count sum_offset = sum_size & 0x1;
			if ((sum_size >>= 1) > 0) {
				cualgo_naive_sum << <1024, 128 >> >(b, sum_size, sum_size + sum_offset, group_size, sum_size * group_num);
				// hipDeviceSynchronize(); // auto sync
				sum_size += sum_offset;
				continue;
			}
			break;
		}
		cualgo_cpy_sum << <1024, 128 >> >(o, b, group_num, group_size);
	}
	
	static void quick_sum(real o[], real b[], count gsz, count gnum) {
		// TODO
	}

	void gbuff::sum(gbuff &b) {
		if (b.size() % size() != 0) {
			INVALID_PARAMETER_FAIL();
		}

		naive_sum(gpubuf(), b.gpubuf(), b.size() / size(), size());
	}

}
