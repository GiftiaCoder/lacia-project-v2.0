#include "hip/hip_runtime.h"

#include "gbuff.h"
#include "gbuff_defs.cuh"

namespace lacia {
	
	__global__ void KERNEL_FUNC_NAME(cpy_sum_result)(real out[], real sum_buf[], count group_num, count group_size) {
		CREATE_TID(group_num)
			out[tid] = sum_buf[tid * group_size];
		DESTORY_TID()
	}

	/*
	 * if group size is even, sum_offset = sum_size
	 * else, sum_offset = sum_size + 1
	*/
	__global__ void KERNEL_FUNC_NAME(naive_sum)(real sum_buf[], count sum_size, count sum_offset, count group_size, count calscale) {
		CREATE_TID(calscale)
			count idx = ((tid / sum_size) * group_size) + (tid % sum_size);
			sum_buf[idx] += sum_buf[idx + sum_offset];
		DESTORY_TID()
	}
	static void naive_sum(real out[], real in[], count group_size, count group_num) {
		count sum_size = group_size;
		while (true) {
			count sum_offset = sum_size & 0x1;
			if ((sum_size >>= 1) > 0) {
				CALL_KERNEL_FUNC(naive_sum, in, sum_size, sum_size + sum_offset, group_size, sum_size * group_num);
				// hipDeviceSynchronize(); // auto sync
				sum_size += sum_offset;
				continue;
			}
			break;
		}
		CALL_KERNEL_FUNC(cpy_sum_result, out, in, group_num, group_size);
	}
	
	static void quick_sum(real o[], real b[], count gsz, count gnum) {
		// TODO
	}

	void gbuff::sum(gbuff &sum_buf) {
		CHECK_ASSERT(sum_buf.size() % size() == 0, INVALID_PARAMETER_FAIL);
		naive_sum(gpubuf(), sum_buf.gpubuf(), sum_buf.size() / size(), size());
	}

}
