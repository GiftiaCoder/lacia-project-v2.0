#include "hip/hip_runtime.h"

#include "gbuff.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <hip/device_functions.h>

namespace lacia {

	__global__ void cualgo_inc_num(real o[], real v, count len) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < len) {
			o[tid] += v;
			tid += tnm;
		}
	}
	void gbuff::inc(real v) {
		cualgo_inc_num << <1024, 128 >> >(gpubuf(), v, size());
	}

	__global__ void cualgo_inc_vec(real o[], real a[], count len) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < len) {
			o[tid] += a[tid];
			tid += tnm;
		}
	}
	void gbuff::inc(gbuff &b) {
		if (size() != b.size()) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_inc_vec << <1024, 128 >> >(gpubuf(), b.gpubuf(), size());
	}

	__global__ void cualgo_add_num(real o[], real a[], real v, count len) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < len) {
			o[tid] = a[tid] + v;
			tid += tnm;
		}
	}
	void gbuff::add(gbuff &b, real v) {
		if (size() != b.size()) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_add_num << <1024, 128 >> >(gpubuf(), b.gpubuf(), v, size());
	}

	__global__ void cualgo_add_vec(real o[], real a[], real b[], count len) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < len) {
			o[tid] = a[tid] + b[tid];
			tid += tnm;
		}
	}
	void gbuff::add(gbuff &b1, gbuff &b2) {
		if (size() != b1.size() || size() != b2.size()) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_add_vec << <1024, 128 >> >(gpubuf(), b1.gpubuf(), b2.gpubuf(), size());
	}

}

