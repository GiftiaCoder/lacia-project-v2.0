#include "hip/hip_runtime.h"

#include "gbuff.h"
#include "gbuff_defs.cuh"

namespace lacia {

#define DEF_GBUFF_DOT_OPS(_name_, _ops_) \
	__global__ void KERNEL_FUNC_NAME(_name_)(real out[], real dot, count calscale) { \
		CREATE_TID(calscale) \
			out[tid] _ops_##= dot; \
		DESTORY_TID() \
	} \
	__global__ void KERNEL_FUNC_NAME(_name_)(real out[], real in[], real dot, count calscale) { \
		CREATE_TID(calscale) \
			out[tid] = in[tid] _ops_ dot; \
		DESTORY_TID() \
	} \
	void gbuff::_name_(real dot) { \
		CALL_KERNEL_FUNC(_name_, gpubuf(), dot, size()); \
	} \
	void gbuff::_name_(gbuff &in, real dot) { \
		CHECK_ASSERT(size() == in.size(), INVALID_PARAMETER_FAIL); \
		CALL_KERNEL_FUNC(_name_, gpubuf(), in.gpubuf(), dot, size()); \
	}
#define DEF_GBUFF_1D_OPS(_name_, _ops_) \
	__global__ void KERNEL_FUNC_NAME(_name_)(real out[], real in[], count calscale) { \
		CREATE_TID(calscale) \
			out[tid] _ops_##= in[tid]; \
		DESTORY_TID() \
	} \
	__global__ void KERNEL_FUNC_NAME(_name_)(real out[], real in1[], real in2[], count calscale) { \
		CREATE_TID(calscale) \
			out[tid] = in1[tid] _ops_ in2[tid]; \
		DESTORY_TID() \
	} \
	void gbuff::_name_(gbuff &in) { \
		CHECK_ASSERT(size() == in.size(), INVALID_PARAMETER_FAIL); \
		CALL_KERNEL_FUNC(_name_, gpubuf(), in.gpubuf(), size()); \
	} \
	void gbuff::_name_(gbuff &in1, gbuff &in2) { \
		CHECK_ASSERT((size() == in1.size()) && (size() == in2.size()), INVALID_PARAMETER_FAIL); \
		CALL_KERNEL_FUNC(_name_, gpubuf(), in1.gpubuf(), in2.gpubuf(), size()); \
	}
#define DEF_GBUFF_NORMAL_OPS(_name_, _ops_) \
	DEF_GBUFF_DOT_OPS(_name_, _ops_) \
	DEF_GBUFF_1D_OPS(_name_, _ops_)

	DEF_GBUFF_NORMAL_OPS(plus, +)
	DEF_GBUFF_NORMAL_OPS(minus, -)
	DEF_GBUFF_NORMAL_OPS(multiple, *)
	DEF_GBUFF_NORMAL_OPS(divide, / )

}
