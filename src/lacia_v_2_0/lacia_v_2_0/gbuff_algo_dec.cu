#include "hip/hip_runtime.h"

#include "gbuff.h"

#include <hip/hip_runtime.h>

namespace lacia {

	__global__ void cualgo_dec_num(real o[], real v, count len) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < len) {
			o[tid] -= v;
			tid += tnm;
		}
	}
	void gbuff::dec(real v) {
		cualgo_dec_num << <1024, 128 >> >(gpubuf(), v, size());
	}

	__global__ void cualgo_dec_vec(real o[], real a[], count len) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < len) {
			o[tid] -= a[tid];
			tid += tnm;
		}
	}
	void gbuff::dec(gbuff &b) {
		if (size() != b.size()) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_dec_vec << <1024, 128 >> >(gpubuf(), b.gpubuf(), size());
	}

	__global__ void cualgo_minus_num(real o[], real a[], real v, count len) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < len) {
			o[tid] = a[tid] - v;
			tid += tnm;
		}
	}
	void gbuff::minus(gbuff &b, real v) {
		if (size() != b.size()) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_minus_num << <1024, 128 >> >(gpubuf(), b.gpubuf(), v, size());
	}

	__global__ void cualgo_minus_vec(real o[], real a[], real b[], count len) {
		count tnm = blockDim.x * gridDim.x;
		count tid = blockIdx.x * blockDim.x + threadIdx.x;
		while (tid < len) {
			o[tid] = a[tid] - b[tid];
			tid += tnm;
		}
	}
	void gbuff::minus(gbuff &b1, gbuff &b2) {
		if (size() != b1.size() || size() != b2.size()) {
			INVALID_PARAMETER_FAIL();
		}
		cualgo_minus_vec << <1024, 128 >> >(gpubuf(), b1.gpubuf(), b2.gpubuf(), size());
	}

}
